#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2021-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: LicenseRef-NvidiaProprietary
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdint.h>

#include "gstnvvideotestsrc.h"
#include "patterns.h"

#define CUDA_CALLABLE __host__ __device__

#define GREY       ColorRGB(0.750, 0.750, 0.750)
#define YELLOW     ColorRGB(0.750, 0.750, 0.000)
#define CYAN       ColorRGB(0.000, 0.750, 0.750)
#define GREEN      ColorRGB(0.000, 0.750, 0.000)
#define MAGENTA    ColorRGB(0.750, 0.000, 0.750)
#define RED        ColorRGB(0.750, 0.000, 0.000)
#define BLUE       ColorRGB(0.000, 0.000, 0.750)
#define OXFORD     ColorRGB(0.000, 0.129, 0.298)
#define WHITE      ColorRGB(1.000, 1.000, 1.000)
#define VIOLET     ColorRGB(0.196, 0.000, 0.416)
#define SUPERBLACK ColorRGB(0.035, 0.035, 0.035)
#define BLACK      ColorRGB(0.075, 0.075, 0.075)
#define DARKGREY   ColorRGB(0.114, 0.114, 0.114)

struct ColorYUV
{
    CUDA_CALLABLE ColorYUV() : y(0), u(0), v(0) {}
    CUDA_CALLABLE ColorYUV(double _y, double _u, double _v) : y(_y), u(_u), v(_v) {}

    double y, u, v;
};

struct ColorRGB
{
    CUDA_CALLABLE ColorRGB() : r(0), g(0), b(0) {}
    CUDA_CALLABLE ColorRGB(double _r, double _g, double _b) : r(_r), g(_g), b(_b) {}

    CUDA_CALLABLE ColorYUV toYUV() const
    {
        double y = (( 0.21260 * r) + ( 0.71520 * g) + ( 0.07220 * b));
        double u = ((-0.114572 * r) + (-0.385428 * g) + ( 0.5 * b)) * 1.02283 + 128;
        double v = (( 0.5 * r) + (-0.454153 * g) + (-0.045847 * b)) * 1.02283 + 128;
        return ColorYUV(y, u, v);
    }

    double r, g, b;
};

__device__
static void write_color(NvBufSurfaceParams *surf, int x, int y, const ColorRGB &rgb)
{
    switch (surf->colorFormat)
    {
        case NVBUF_COLOR_FORMAT_RGBA:
        {
            uint8_t *p = (uint8_t*)surf->dataPtr + (y * surf->pitch) + (x * 4);
            p[0] = (int)(rgb.r * 255);
            p[1] = (int)(rgb.g * 255);
            p[2] = (int)(rgb.b * 255);
            p[3] = 255;
            break;
        }
        case NVBUF_COLOR_FORMAT_NV12_709:
        {
            ColorYUV yuv = rgb.toYUV();
            uint8_t *p = (uint8_t*)surf->dataPtr + (y * surf->pitch) + x;
            *p = (int)(yuv.y * 255);

            if (x % 2 == 0 && y % 2 == 0)
            {
                uint8_t *uv = (uint8_t*)surf->dataPtr +
                    surf->planeParams.offset[1] +
                    ((y / 2) * surf->planeParams.pitch[1]) + x;
                uv[0] = (int)(yuv.u * 255);
                uv[1] = (int)(yuv.v * 255);
            }
            break;
        }
        case NVBUF_COLOR_FORMAT_YUV420_709:
        {
            ColorYUV yuv = rgb.toYUV();
            uint8_t *p = (uint8_t*)surf->dataPtr + (y * surf->pitch) + x;
            *p = (int)(yuv.y * 255);

            if (x % 2 == 0 && y % 2 == 0)
            {
                uint8_t *u = (uint8_t*)surf->dataPtr +
                    surf->planeParams.offset[1] +
                    ((y / 2) * surf->planeParams.pitch[1]) + (x / 2);
                *u = (int)(yuv.u * 255);
                uint8_t *v = (uint8_t*)surf->dataPtr +
                    surf->planeParams.offset[2] +
                    ((y / 2) * surf->planeParams.pitch[2]) + (x / 2);
                *v = (int)(yuv.v * 255);
            }
            break;
        }
        case NVBUF_COLOR_FORMAT_UYVP:
        case NVBUF_COLOR_FORMAT_UYVP_ER:
        {
            // UYVP is 10-bit 4:2:2 packed format
            // Each group of 2 pixels (U|Y, V|Y) is packed into 5 bytes
            // Layout: U0(10) Y0(10) V0(10) Y1(10) = 40 bits = 5 bytes
            // We need to handle 2 pixels at a time

            if (x % 2 == 0) // Only process even pixels, handle pairs
            {
                ColorYUV yuv = rgb.toYUV();

                // Calculate byte position for this pixel pair
                int pair_index = x / 2;
                int byte_offset = (y * surf->pitch) + (pair_index * 5);
                uint8_t *p = (uint8_t*)surf->dataPtr + byte_offset;

                // Convert 8-bit YUV to 10-bit (multiply by 4)
                uint16_t u10 = ((uint16_t)(yuv.u * 255)) << 2;
                uint16_t y0_10 = ((uint16_t)(yuv.y * 255)) << 2;
                uint16_t v10 = ((uint16_t)(yuv.v * 255)) << 2;
                uint16_t y1_10 = y0_10; // For test pattern, use same Y for both pixels

                // Pack into 5 bytes: U0(10) Y0(10) V0(10) Y1(10)
                // Byte 0: U0[9:2]
                p[0] = (u10 >> 2) & 0xFF;
                // Byte 1: U0[1:0] Y0[9:4]
                p[1] = ((u10 & 0x03) << 6) | ((y0_10 >> 4) & 0x3F);
                // Byte 2: Y0[3:0] V0[9:6]
                p[2] = ((y0_10 & 0x0F) << 4) | ((v10 >> 6) & 0x0F);
                // Byte 3: V0[5:0] Y1[9:8]
                p[3] = ((v10 & 0x3F) << 2) | ((y1_10 >> 8) & 0x03);
                // Byte 4: Y1[7:0]
                p[4] = y1_10 & 0xFF;
            }
            break;
        }
        case NVBUF_COLOR_FORMAT_BGRA64_LE:
        {
            // BGRA64_LE is 64 bits per pixel (16 bits per component, little-endian)
            // Layout: B(16) G(16) R(16) A(16) in little-endian order
            uint16_t *p = (uint16_t*)((uint8_t*)surf->dataPtr + (y * surf->pitch) + (x * 8));

            // Convert from 0.0-1.0 range to 0-65535 range (16-bit)
            p[0] = (uint16_t)(rgb.b * 65535);  // Blue
            p[1] = (uint16_t)(rgb.g * 65535);  // Green
            p[2] = (uint16_t)(rgb.r * 65535);  // Red
            p[3] = 65535;                       // Alpha (fully opaque)
            break;
        }
        case NVBUF_COLOR_FORMAT_RGBA_10_10_10_2_709:
        case NVBUF_COLOR_FORMAT_RGBA_10_10_10_2_2020:
        {
            // RGB10A2_LE is 32 bits per pixel (10 bits for R,G,B and 2 bits for A)
            // Layout in little-endian: R[9:0] G[9:0] B[9:0] A[1:0]
            uint32_t *p = (uint32_t*)((uint8_t*)surf->dataPtr + (y * surf->pitch) + (x * 4));

            // Convert from 0.0-1.0 range to 0-1023 range (10-bit)
            uint32_t r10 = (uint32_t)(rgb.r * 1023) & 0x3FF;
            uint32_t g10 = (uint32_t)(rgb.g * 1023) & 0x3FF;
            uint32_t b10 = (uint32_t)(rgb.b * 1023) & 0x3FF;
            uint32_t a2 = 3; // Alpha (fully opaque - 2 bits)

            // Pack into 32-bit word: A[1:0] B[9:0] G[9:0] R[9:0]
            *p = (a2 << 30) | (b10 << 20) | (g10 << 10) | r10;
            break;
        }
    }
}

__global__
void smpte_kernel(NvBufSurfaceParams *surf, int horizontal_offset)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < surf->width * surf->height; i += stride)
    {
        int p_y = i / surf->width;
        int p_x = i - (p_y * surf->width);

        // Positive horizontal_speed moves pattern to the left
        int effective_x = (p_x + horizontal_offset) % surf->width;
        if (effective_x < 0) {
          effective_x += surf->width;
        }
        ColorRGB c;
        if (p_y < 0.67 * surf->height)
        {
            int bar_width = surf->width / 7.0;
            if (effective_x < bar_width)
                c = GREY;
            else if (effective_x < bar_width * 2)
                c = YELLOW;
            else if (effective_x < bar_width * 3)
                c = CYAN;
            else if (effective_x < bar_width * 4)
                c = GREEN;
            else if (effective_x < bar_width * 5)
                c = MAGENTA;
            else if (effective_x < bar_width * 6)
                c = RED;
            else
                c = BLUE;
        }
        else if (p_y < 0.75 * surf->height)
        {
            int bar_width = surf->width / 7.0;
            if (effective_x < bar_width)
                c = BLUE;
            else if (effective_x < bar_width * 2)
                c = BLACK;
            else if (effective_x < bar_width * 3)
                c = MAGENTA;
            else if (effective_x < bar_width * 4)
                c = BLACK;
            else if (effective_x < bar_width * 5)
                c = CYAN;
            else if (effective_x < bar_width * 6)
                c = BLACK;
            else
                c = GREY;
        }
        else
        {
            int bar_width = (surf->width / 7.0 * 5.0) / 4.0;
            if (effective_x < bar_width)
                c = OXFORD;
            else if (effective_x < bar_width * 2)
                c = WHITE;
            else if (effective_x < bar_width * 3)
                c = VIOLET;
            else if (effective_x < bar_width * 4)
                c = BLACK;
            else if (effective_x < (surf->width / 21.0) * 16)
                c = SUPERBLACK;
            else if (effective_x < (surf->width / 21.0) * 17)
                c = BLACK;
            else if (effective_x < (int)(surf->width / 7.0) * 6)
                c = DARKGREY;
            else
                c = BLACK;
        }
        write_color(surf, p_x, p_y, c);
    }
}

extern "C"
void gst_nv_video_test_src_smpte(GstNvVideoTestSrc *src)
{
    // Calculate horizontal offset based on frame number and speed
    int horizontal_offset = src->filled_frames * src->horizontal_speed;

    // Pass the offset to the kernel
    smpte_kernel<<<src->cuda_num_blocks, src->cuda_block_size>>>(src->cuda_surf, horizontal_offset);
}

__device__
static int mandelbrot(double x, double y, int max_iter)
{
    double a = 0, b = 0, asq = 0, bsq = 0;

    int i = 0;
    while (i++ < max_iter)
    {
        b = (a * b) * 2 + y;
        a = asq - bsq + x;
        asq = a * a;
        bsq = b * b;

        if (asq + bsq > 4.0)
            break;
    }

    return i;
}

__global__
void mandelbrot_kernel(NvBufSurfaceParams *surf, ColorRGB *colors, int num_colors,
                       double x_off, double y_off, double scale)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < surf->width * surf->height; i += stride)
    {
        int p_y = i / surf->width;
        int p_x = i - (p_y * surf->width);
        double x = (((double)p_x / surf->width) * 2 - 1) * scale + x_off;
        double y = (((double)p_y / surf->height) * 2 - 1) * scale + y_off;
        ColorRGB &c = colors[mandelbrot(x, y, num_colors)];
        write_color(surf, p_x, p_y, c);
    }
}

extern "C"
void gst_nv_video_test_src_mandelbrot(GstNvVideoTestSrc *src)
{
    // Initialize the static color array on the first call.
    static const int MAX_ITER = 100;
    static ColorRGB *colors = NULL;
    if (!colors)
    {
        ColorRGB host_colors[MAX_ITER + 1];
        for (int i = 0; i <= MAX_ITER; i++)
        {
            float freq = 6.3 / MAX_ITER;
            double r = (sin(freq * i + 5) + 1.0) / 2.0;
            double g = (sin(freq * i + 4) + 1.0) / 2.0;
            double b = (sin(freq * i + 3) + 1.0) / 2.0;
            host_colors[i] = ColorRGB(r, g, b);
        }
        host_colors[MAX_ITER] = ColorRGB(0, 0, 0);

        int size = sizeof(ColorRGB) * (MAX_ITER + 1);
        hipMalloc((void**)&colors, size);
        hipMemcpy(colors, host_colors, size, hipMemcpyHostToDevice);
    }

    const double   MIN_SCALE  = 0.000001;
    const double   MAX_SCALE  = 2.0;
    const double   X_OFF      = -0.734072;
    const double   Y_OFF      =  0.248116;
    const double   log_min    = log(MIN_SCALE);
    const double   log_max    = log(MAX_SCALE);

    // Determine the zoom/scale.
    double interp;
    if (src->animation_mode == GST_NV_VIDEO_TEST_SRC_FRAMES)
    {
        const uint32_t ZOOM_SPEED = 500;
        interp = (src->filled_frames / ZOOM_SPEED) % 2
            ? (src->filled_frames % ZOOM_SPEED) / (double)ZOOM_SPEED
            : (ZOOM_SPEED - (src->filled_frames % ZOOM_SPEED)) / (double)ZOOM_SPEED;
    }
    else
    {
        const uint32_t ZOOM_SPEED = 10000000;
        guint time;
        if (src->animation_mode == GST_NV_VIDEO_TEST_SRC_WALL_TIME)
            time = g_get_real_time();
        else
            time = src->running_time / 1000;
        interp = (time / ZOOM_SPEED) % 2
            ? (time % ZOOM_SPEED) / (double)ZOOM_SPEED
            : (ZOOM_SPEED - (time % ZOOM_SPEED)) / (double)ZOOM_SPEED;
    }
    double log_scale = log_min + (log_max - log_min) * interp;
    double scale = exp(log_scale);

    mandelbrot_kernel<<<src->cuda_num_blocks, src->cuda_block_size>>>(
        src->cuda_surf, colors, MAX_ITER, X_OFF, Y_OFF, scale);
}

__global__
void gradient_kernel(NvBufSurfaceParams *surf)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < surf->width * surf->height; i += stride)
    {
        int p_y = i / surf->width;
        int p_x = i - (p_y * surf->width);
        double x = (double)p_x / surf->width;
        double y = (double)p_y / surf->height;
        ColorRGB c(x, y, (x + y) / 2.0);
        write_color(surf, p_x, p_y, c);
    }
}

extern "C"
void gst_nv_video_test_src_gradient(GstNvVideoTestSrc *src)
{
    gradient_kernel<<<src->cuda_num_blocks, src->cuda_block_size>>>(src->cuda_surf);
}

extern "C"
void gst_nv_video_test_src_cuda_init(GstNvVideoTestSrc *src)
{
    if (src->cuda_surf)
        hipFree(src->cuda_surf);

    // The details of the surface to be rendered into by CUDA are provided each
    // frame by the NvBufSurfaceParams. Since these details are read by the CUDA
    // kernel, we need to copy this struct into CUDA-accessible memory. We
    // allocate CUDA memory for this structure here.
    hipMalloc(&src->cuda_surf, sizeof(NvBufSurfaceParams));
    src->cuda_block_size = 512;
    src->cuda_num_blocks = (src->info.width * src->info.height +
        src->cuda_block_size - 1) / src->cuda_block_size;
}

extern "C"
void gst_nv_video_test_src_cuda_free(GstNvVideoTestSrc *src)
{
    hipFree(src->cuda_surf);
}

extern "C"
void gst_nv_video_test_src_cuda_prepare(
    GstNvVideoTestSrc *src, NvBufSurfaceParams *surf)
{
    // Copy the details of the surface that we're about to render into
    // to the CUDA-accessible copy.
    hipMemcpy(src->cuda_surf, surf, sizeof(NvBufSurfaceParams),
        hipMemcpyHostToDevice);
}
